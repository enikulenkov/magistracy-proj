#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <hipsparse.h>

/**
 * CUDA Kernel Device code
 *
 * Computes dense matrix on vector multiplication. "Naive" implementation.
 */
__global__ void
matr_vector_mult(const double *A, const double *B, double *C, int numElements)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j;

  if (i < numElements)
  {
    double sum = 0;

    for (j = 0; j < numElements; j++)
    {
      sum += A[i * numElements + j] * B[j];
    }

    C[i] = sum;
  }
}

/* Wrapper to unify calling from Fortran and C */
extern "C" void matr_vector_mult_(double *h_A, double *h_B, double *h_C, int *numElements)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    size_t size = *numElements * sizeof(double);

    // Allocate the device input matrix A
    double *d_A = NULL;
    err = hipMalloc((void **)&d_A, (*numElements)*(*numElements)*sizeof(double));

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    double *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    double *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, (*numElements)*(*numElements)*sizeof(double), hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(*numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    matr_vector_mult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, *numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch matr_vector_mult kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device matrix A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

extern "C" void cuda_deinit_()
{
    hipError_t err;

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

extern "C" void matr_vector_mult_sparse(int *h_A_row_indeces, int *h_A_col_indeces, double *h_A_values, int nnz,
    double *h_B, double *h_C, int *numElements)
{
  hipsparseHandle_t cusparse_hndl;
  hipsparseStatus_t cusparse_ret;
  hipsparseMatDescr_t mat_descr;
  hipError_t err = hipSuccess;

  int *d_A_row_indices = NULL;
  int *d_A_col_indices = NULL;
  double *d_A_values = NULL;
  int *csrRowPtr;
  double done = 1;
  double complex_null = 0;
  hipMalloc((void **)&d_A_row_indices, nnz*sizeof(int));
  hipMalloc((void **)&d_A_col_indices, nnz*sizeof(int));
  hipMalloc((void **)&d_A_values, nnz*sizeof(double));

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  double *d_B = NULL;
  err = hipMalloc((void **)&d_B, *numElements*sizeof(double));

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  double *d_C = NULL;
  err = hipMalloc((void **)&d_C, *numElements*sizeof(double));

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  hipMemcpy(d_A_row_indices, h_A_row_indeces, nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_A_col_indices, h_A_col_indeces, nnz*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_A_values, h_A_values, nnz*sizeof(double), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy matrix A from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, *numElements*sizeof(double), hipMemcpyHostToDevice);

  if (err != hipSuccess)
  {
      fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  cusparse_ret = hipsparseCreate(&cusparse_hndl);

  if (cusparse_ret != HIPSPARSE_STATUS_SUCCESS)
  {
    fprintf(stderr, "Failed to create cuSparse context (error code %d)!\n", cusparse_ret);
    exit(EXIT_FAILURE);
  }

  cusparse_ret = hipsparseCreateMatDescr(&mat_descr);

  if (cusparse_ret != HIPSPARSE_STATUS_SUCCESS)
  {
    fprintf(stderr, "Failed to create cuSparse matrix description (error code %d)!\n", cusparse_ret);
    exit(EXIT_FAILURE);
  }

  hipsparseSetMatType(mat_descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(mat_descr, HIPSPARSE_INDEX_BASE_ZERO);

  /* exercise conversion routines (convert matrix from COO 2 CSR format) */
  hipMalloc((void**)&csrRowPtr,(*numElements+1)*sizeof(csrRowPtr[0]));

  cusparse_ret = hipsparseXcoo2csr(cusparse_hndl, d_A_col_indices, nnz, *numElements,
                             csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO); 

  if (cusparse_ret != HIPSPARSE_STATUS_SUCCESS)
  {
    fprintf(stderr, "Failed to make a conversion COO->CSR (error code %d)!\n", cusparse_ret);
    exit(EXIT_FAILURE);
  }

  hipMemset(d_C, 0, nnz*sizeof(double));
  cusparse_ret = hipsparseDcsrmv(cusparse_hndl,HIPSPARSE_OPERATION_NON_TRANSPOSE,*numElements, *numElements,
                           done, mat_descr, d_A_values, csrRowPtr, d_A_row_indices, 
                           d_B, complex_null, d_C);

  if (cusparse_ret != HIPSPARSE_STATUS_SUCCESS)
  {
    fprintf(stderr, "Failed to make cusparse mult (error code %d)!\n", cusparse_ret);
    exit(EXIT_FAILURE);
  }

  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, *numElements*sizeof(double), hipMemcpyDeviceToHost);
}
